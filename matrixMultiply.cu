#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

// Number of rows for matrix A
#define row1 64

// Number of cols for matrix A
#define col1 8

// Number of rows for matrix B
#define row2 8

// Number of cols for matrix B
#define col2 64

void matrixMultiplyCPU(int *A, int *B, int *C, int x, int y, int z)
{
    // CPU Matrix Multiplication
    for (int i = 0; i < x; i++)
    {
        for (int j = 0; j < z; j++)
        {
            for (int k = 0; k < y; k++)
            {
                C[i*z+j] += A[i*y+k] * B[k*z+j];
                //C[i][j] += A[i][k] * B[k][j];
            }
        }
    }
}

__global__ void matrixMultiplyShared(int* a, int *b, int *c)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    int k = threadIdx.x; 
    __shared__ int p[col1];

    c[col2*y+x] = 0;
    p[k] = a[col1*y+k]*b[col2*k+x];

    __syncthreads();

    for(int i = 0; i < col1; i++)
    {
        c[col2*y+x] = c[col2*y+x]+p[i];
    }
}

void printMatrix(int *A, int numRow, int numCol)
{
    for (int i = 0; i < numRow; i++) 
    {
        for (int j = 0; j < numCol; j++) 
        {
            cout << A[i*numCol+j] << " ";
        }
        cout << endl;
    }
}

int main()
{
    int *A = new int[row1*col1];
    int *B = new int[row2*col2];
    int *C = new int[row1*col2];
    int *res = new int[row1*col2];
    int *a_d, *b_d, *c_d;

    // Load A with data
    for (int i = 0; i < row1; i++) {
        for (int j = 0; j < col1; j++) {
            //A[i][j] = i+1;
            A[i*col1+j] = i+1;
        }
    }

    // Print A
    //printMatrix(A, row1, row2);
   
    // Load B with data
    for (int i = 0; i < row2; i++) {
        for (int j = 0; j < col2; j++) {
            B[i*col2+j] = j+1;
        }
    }

    // Print B
    //printMatrix(B, row2, col2);

    // Load C with data
    for (int i = 0; i < row1; i++) {
        for (int j = 0; j < col2; j++) {
            //C[i][j] = 0;
            C[i*col2+j] = 0;
        }
    }

    // Print C
    //printMatrix(C, row1, col2);
    
    
    hipMalloc((void **) &a_d, row1*col1*sizeof(int));
    hipMalloc((void **) &b_d, row2*col2*sizeof(int));
    hipMalloc((void **) &c_d, row1*col2*sizeof(int));

    hipMemcpy(a_d, A, row1*col1*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_d, B, row2*col2*sizeof(int), hipMemcpyHostToDevice);

    dim3 grid(col2, row1);
	
    // Time start
    float gpuTimeTaken = 0.0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    // Call Kernel Function
    matrixMultiplyShared<<<grid, col1>>>(a_d, b_d, c_d);
    hipMemcpy(res, c_d, row1*col2*sizeof(int), hipMemcpyDeviceToHost);
    
    // Time end
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTimeTaken, start, stop);
    cout << "Time taken by GPU Matrix Multiplication in ms: " << gpuTimeTaken << endl;
   
    // CPU Matrix Multiplication
    matrixMultiplyCPU(A, B, C, row1, row2, col2);

    // Compare CPU Matrix Multiply with GPU Matrix Multiply    
    bool correct = false;
    for (int i = 0; i < row1; i++)
    {
	for (int j = 0; j < col2; j++)
	{
	    if (C[i*col2+j] == res[i*col2+j])
	    {
		correct = true;
	    }
	}
    }		
    if (correct)
    {
	cout << "CPU Matrix Multiplication is same as GPU Matrix Multiplication" << endl;
    }
    else
    {
	cout << "Results not the same" << endl;	
    }

    // Free Memory
    delete[] A;
    delete[] B;
    delete[] C;
    delete[] res;
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);

    return 0;
}
