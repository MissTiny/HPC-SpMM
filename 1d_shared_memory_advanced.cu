#include "hip/hip_runtime.h"
#include <iostream>
#include<ctime>
#include<stdio.h>
using namespace std;


void CPU_matrix_multiplication(int *A, int *B, int *C, int N){
    for(int m=0;m<N;m++){
        for(int n=0;n<N;n++){
            for(int k=0;k<N;k++){
                C[m*N+n] +=A[m*N+k] * B[k*N+n];
            }
        }
    }
}


__global__ void CUDA_matrix_multiplication(int *A, int *B, int *C, int N){
    
    __shared__ int smem_c[64][64];
    __shared__ int smem_a[64][8];
    __shared__ int smem_b[8][64];

    int c = blockIdx.x * 64;
    int r = blockIdx.y *64;
    smem_c[threadIdx.y*2][threadIdx.x*2] =0;
    smem_c[threadIdx.y*2][threadIdx.x*2+1]=0;
    smem_c[threadIdx.y*2+1][threadIdx.x*2]=0;
    smem_c[threadIdx.y*2+1][threadIdx.x*2+1]=0;
    for (int kk=0;kk<N;kk+=8){
        for (int i=threadIdx.x+blockDim.x*threadIdx.y; i<64*8; i+=blockDim.x*blockDim.y){
            int k = kk+i/64;
            int rt = r + i%64;
            int ct = c + i%64;
            smem_a[i%64][i/64] = A[rt*N+k];
            smem_b[i/64][i%64] = B[k*N+ct];
        }
    
        __syncthreads();
        	
        for (int n=0; n<(8);n++){
            smem_c[threadIdx.y*2][threadIdx.x*2] += smem_a[threadIdx.y*2][n] * smem_b[n][threadIdx.x*2];
            smem_c[threadIdx.y*2][threadIdx.x*2+1] += smem_a[threadIdx.y*2][n] * smem_b[n][threadIdx.x*2+1];
            smem_c[threadIdx.y*2+1][threadIdx.x*2] += smem_a[threadIdx.y*2+1][n] * smem_b[n][threadIdx.x*2];
            smem_c[threadIdx.y*2+1][threadIdx.x*2+1] += smem_a[threadIdx.y*2+1][n] * smem_b[n][threadIdx.x*2+1];
        }

        __syncthreads();
    }

    //if ((c+threadIdx.x) < N && (r+threadIdx.y)<N){
        C[((blockIdx.y*64 + threadIdx.y*2)*N) + (blockIdx.x * 64)+threadIdx.x*2] = smem_c[threadIdx.y*2][threadIdx.x*2];
        C[((blockIdx.y*64 + threadIdx.y*2+1)*N) + (blockIdx.x * 64)+threadIdx.x*2] = smem_c[threadIdx.y*2+1][threadIdx.x*2];
        C[((blockIdx.y*64 + threadIdx.y*2)*N) + (blockIdx.x * 64)+threadIdx.x*2+1] = smem_c[threadIdx.y*2][threadIdx.x*2+1];
        C[((blockIdx.y*64 + threadIdx.y*2+1)*N) + (blockIdx.x * 64)+threadIdx.x*2+1] = smem_c[threadIdx.y*2+1][threadIdx.x*2+1];
    //}




}


int main()
{
    //init established
    int N,size, memSize; // M for row. K reduction, N for column
    N=4096;
    size = N*N;
    memSize = size*sizeof(int);

    int *A = new int[size];
    int *B = new int[size];
    int *C_1 = new int[size];
    int *C_2 = new int[size];
    int *d_A;
    int *d_B;
    int *d_C;
    //random init A
    for (int m=0; m<N;m++){
        for(int k=0; k<N; k++){
            A[m*N+k] = m+1;
        }
    }

    //random init B
    for (int k=0; k<N;k++){
        for (int n=0;n<N;n++){
            B[k*N+n] = k+1;
        }
    }

    for (int m=0; m<N;m++){
        for (int n=0;n<N;n++){
            C_1[m*N+n] = 0;
            C_2[m*N+n] = 0;
        }
    }
    cout << "A: "<<N<<" x " <<N <<", B: " <<N<<" x " <<N <<", C: "<<N<<" x " <<N<<endl;
    cout << "CPU Multiplication: Start" <<endl;
    
    CPU_matrix_multiplication(A,B,C_1,N);
    cout<<"CPU Multiplication: Finish"<<endl;
    hipMalloc((void **) &d_A, memSize);
    hipMemcpy(d_A, A,memSize,hipMemcpyHostToDevice);

    hipMalloc((void **) &d_B, memSize);
    hipMemcpy(d_B, B,memSize,hipMemcpyHostToDevice);

    hipMalloc((void **) &d_C,memSize);
    hipMemcpy(d_C, C_2,memSize,hipMemcpyHostToDevice);

    //launch kernel

    dim3 nblocks((N+64-1)/64,(N+64-1)/64);
    dim3 nthreads(32,32);
    cout << "CUDA Multiplication: Start" <<endl;
    float time = 0.0;
    hipEvent_t begin,end;
    hipEventCreate(&begin);
    hipEventCreate(&end);
    hipEventRecord(begin,0);
    CUDA_matrix_multiplication<<<nblocks,nthreads>>>(d_A, d_B, d_C,N);
    hipMemcpy(C_2,d_C,memSize,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(end,0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time,begin,end);
    cout << "CUDA Multiplication: It took me " << time << " seonds." <<endl;
   
    bool error = false;
    for (int i=0; i<size;i++){
	
        if(C_1[i] != C_2[i]){
            error = true;
            cout<<"Error: the cuda is not correct at i= ["<< i <<"] place" <<endl;
        }
	
    }
    if(!error){
        cout<<"Correct: the cuda result is the same as CPU result"<<endl;
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
