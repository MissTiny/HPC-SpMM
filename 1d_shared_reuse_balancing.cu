#include "hip/hip_runtime.h"
#include <iostream>
#include<ctime>
#include<stdio.h>
#include<fstream>
using namespace std;


__global__ void CUDA_matrix_multiplication(double *A, double *B, double *C, int N){
    
    __shared__ double smem_c[64][64];
    __shared__ double smem_a[64][8];
    __shared__ double smem_b[8][64];

    int c = blockIdx.x * 64;
    int r = blockIdx.y *64;
    smem_c[threadIdx.y*2][threadIdx.x*2] =0;
    smem_c[threadIdx.y*2][threadIdx.x*2+1]=0;
    smem_c[threadIdx.y*2+1][threadIdx.x*2]=0;
    smem_c[threadIdx.y*2+1][threadIdx.x*2+1]=0;
    for (int kk=0;kk<N;kk+=8){
        for (int i=threadIdx.x+blockDim.x*threadIdx.y; i<64*8; i+=blockDim.x*blockDim.y){
            int k = kk+i/64;
            int rt = r + i%64;
            int ct = c + i%64;
            smem_a[i%64][i/64] = A[rt*N+k];
            smem_b[i/64][i%64] = B[k*N+ct];
        }
    
        __syncthreads();
        	
        for (int n=0; n<(8);n++){
            smem_c[threadIdx.y*2][threadIdx.x*2] += smem_a[threadIdx.y*2][n] * smem_b[n][threadIdx.x*2];
            smem_c[threadIdx.y*2][threadIdx.x*2+1] += smem_a[threadIdx.y*2][n] * smem_b[n][threadIdx.x*2+1];
            smem_c[threadIdx.y*2+1][threadIdx.x*2] += smem_a[threadIdx.y*2+1][n] * smem_b[n][threadIdx.x*2];
            smem_c[threadIdx.y*2+1][threadIdx.x*2+1] += smem_a[threadIdx.y*2+1][n] * smem_b[n][threadIdx.x*2+1];
        }

        __syncthreads();
    }

    //if ((c+threadIdx.x) < N && (r+threadIdx.y)<N){
        C[((blockIdx.y*64 + threadIdx.y*2)*N) + (blockIdx.x * 64)+threadIdx.x*2] = smem_c[threadIdx.y*2][threadIdx.x*2];
        C[((blockIdx.y*64 + threadIdx.y*2+1)*N) + (blockIdx.x * 64)+threadIdx.x*2] = smem_c[threadIdx.y*2+1][threadIdx.x*2];
        C[((blockIdx.y*64 + threadIdx.y*2)*N) + (blockIdx.x * 64)+threadIdx.x*2+1] = smem_c[threadIdx.y*2][threadIdx.x*2+1];
        C[((blockIdx.y*64 + threadIdx.y*2+1)*N) + (blockIdx.x * 64)+threadIdx.x*2+1] = smem_c[threadIdx.y*2+1][threadIdx.x*2+1];
    //}




}


int main()
{
    double *d_A;
    double *d_B;
    double *d_C;
	//Loading A sparse matrix
	ifstream fin1("data_mtx/A0_dense_float.mtx");
	//ifstream fin("test2.mtx");
	//while (fin.peek() == '%') fin.ignore(2048, '\n');

	int M_A, K_A, L_A;
	fin1 >> M_A >> K_A >> L_A;

    int line_notsame = 12;
    int size_A = (line_notsame+1)*K_A;
    int memSize_A = size_A *sizeof(double);
	//B as a normal Matrix
	cout << "start A as normal matrix construction\n";
	//int* V_B = new int[K_B][N_B];
    double* V_A = new double[M_A *K_A];
    double* new_A = new double[(line_notsame+1)*K_A];
	for (int i = 0; i < L_A; i++) {
		int row_A, col_A;
		double value_A;
		fin1 >> row_A >> col_A >> value_A;
		//ROW_INDEX_B[i] = row_B;
		//COL_INDEX_B[i] = col_B;
        V_A[row_A * K_A + col_A] = value_A;
        if(row_A<line_notsame){
            new_A[row_A * K_A + col_A] = value_A;
        }
	}
	cout << "A matrix construction finished\n";
	fin1.close();
	//matrix construct
    //Loading B
    ifstream fin2("data_mtx/B_sparse90_loadbalancing.mtx");
	//ifstream fin("test2.mtx");
	//while (fin.peek() == '%') fin.ignore(2048, '\n');
	int K_B, N_B, L_B;
	fin2 >> K_B >> N_B >> L_B;
    int size_B = K_B*N_B;
    int memSize_B = size_B *sizeof(double);
	//A SpMM Matrix Validation Check
	if (K_A != K_B){
		cout<< "Matrix A and Matrix B can not be multipled together";
		return 1;
	}
	//B as a normal Matrix
	cout << "start B as normal matrix construction\n";
	//int* V_B = new int[K_B][N_B];
	double* V_B = new double[K_B*N_B];
	for (int i = 0; i < L_B; i++) {
		int row_B, col_B;
		double value_B;
		fin2 >> row_B >> col_B >> value_B;
		V_B[row_B*N_B+ col_B] = value_B;
	}
	cout << "B matrix construction finished\n";
	fin2.close();

    int size_C = (line_notsame+1)*N_B;
    int memSize_C = size_C *sizeof(double);
    double* V_C = new double[(line_notsame+1)*N_B];
    for (int m=0; m<(line_notsame+1);m++){
        for (int n=0;n<N_B;n++){
            V_C[m*N_B+n] = 0;
        }
    }
    hipMalloc((void **) &d_A, memSize_A);
    hipMemcpy(d_A, new_A,memSize_A,hipMemcpyHostToDevice);

    hipMalloc((void **) &d_B, memSize_B);
    hipMemcpy(d_B, V_B,memSize_B,hipMemcpyHostToDevice);

    hipMalloc((void **) &d_C,memSize_C);
    hipMemcpy(d_C, V_C,memSize_C,hipMemcpyHostToDevice);

    //launch kernel

    dim3 nblocks((M_A+64-1)/64,(N_B+64-1)/64);
    dim3 nthreads(32,32);
    cout << "CUDA Multiplication: Start" <<endl;
    float time = 0.0;
    hipEvent_t begin,end;
    hipEventCreate(&begin);
    hipEventCreate(&end);
    hipEventRecord(begin,0);
    CUDA_matrix_multiplication<<<nblocks,nthreads>>>(d_A, d_B, d_C,K_A);
    hipMemcpy(V_C,d_C,memSize_C,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipEventRecord(end,0);
    hipEventSynchronize(end);
    hipEventElapsedTime(&time,begin,end);
    cout << "CUDA Multiplication: It took me " << time << " milliseconds." <<endl;


    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
